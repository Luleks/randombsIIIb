#include "hip/hip_runtime.h"
/*

UNFINISHED

*/

#include <stdio.h>
#include <stdlib.h>


__global__ void kernel(int* a, int* b, int* n) {
    if (threadIdx.x >= *n || threadIdx.y >= *n)
        return;
    
    int tid

    b[threadIdx.x][threadIdx.y] = a[threadIdx.x][threadIdx.y];
}

__host__ void initAndCall(int* a, int* b, int n) {
    int* dev_a, *dev_b, *dev_n;
    hipMalloc((void**)&dev_a, n * n * sizeof(int));
    hipMalloc((void**)&dev_b, n * n * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMemcpy(dev_a, a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(256, 256);
    dim3 gridDim(256, 256);

    kernel<<<gridDim, blockDim>>>(dev_a, dev_b);

    hipMemcpy(b, dev_b, n * n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {

}