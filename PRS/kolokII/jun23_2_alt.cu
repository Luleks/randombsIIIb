#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256

__global__ void get_diag_min(int* a, int* b, int n) {
    __shared__ int partial_min[BLOCK_SIZE];
    partial_min[threadIdx.x] = INT_MAX;

    int row = threadIdx.x + blockIdx.x * blockDim.x;
    while (row < n) {
        int diag_idx = row * n + row;
        partial_min[threadIdx.x] = min(a[diag_idx], partial_min[threadIdx.x]);
        row += blockDim.x * gridDim.x;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_min[threadIdx.x] = min(partial_min[threadIdx.x], partial_min[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        b[blockIdx.x] = partial_min[0];
    }
}

__host__ int get_min(int* mat, int n) {
    int* dev_a, *dev_b;
    hipMalloc((void**)&dev_a, n * n * sizeof(int));
    hipMalloc((void**)&dev_b, GRID_SIZE * sizeof(int));
    hipMemcpy(dev_a, mat, n * n * sizeof(int), hipMemcpyHostToDevice);

    get_diag_min<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, n);

    int* partial_mins = (int*)malloc(GRID_SIZE * sizeof(int));
    hipMemcpy(partial_mins, dev_b, GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    int reduced_min = INT_MAX;
    for (int i = 0; i < GRID_SIZE; ++i) {
        reduced_min = min(reduced_min, partial_mins[i]);
    }

    free(partial_mins);
    hipFree(dev_a);
    hipFree(dev_b);

    return reduced_min;
}

int main() {
    int N = 720;
    
    int* matrix = (int*)malloc(N * N * sizeof(int));
    int actual_min = INT_MAX;

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            matrix[i * N + j] = rand();
            if (i == j) actual_min = min(actual_min, matrix[i * N + j]);
        }
    }

    int reduced_min = get_min(matrix, N);

    printf("%d == %d\n", actual_min, reduced_min);

    free(matrix);
    return 0;
}
