
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 4
#define GRID_SIZE 3


__global__ void kernel(int* a, int* b, int* n) {
    __shared__ int local_a[BLOCK_SIZE + 2];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < *n) {
        local_a[threadIdx.x] = a[tid];

        if (tid >= *n - 2) {
            return;
        }
        if (threadIdx.x < 2) {
            local_a[threadIdx.x + blockDim.x] = a[tid + blockDim.x];
        }
        __syncthreads();

        int temp = (local_a[threadIdx.x] + local_a[threadIdx.x + 1] + local_a[threadIdx.x + 2]) /
        (local_a[threadIdx.x] * local_a[threadIdx.x + 1] * local_a[threadIdx.x + 2]);
        b[tid] = temp;

        tid += BLOCK_SIZE * GRID_SIZE;
    }
}

__host__ void initAndCall(int* a, int* b, int n) {
    int* dev_a, *dev_b, *dev_n;
    hipMalloc((void**)&dev_a, (n + 2) * sizeof(int));
    hipMalloc((void**)&dev_b, (n - 2) * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_n);

    hipMemcpy(b, dev_b, (n - 2) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);    

}

int main() {
    int n = 33;
    
    int* a = (int*)malloc(n * sizeof(int));
    int* b = (int*)malloc((n - 2) * sizeof(int));

    for(int i = 0; i < n; ++i) {
        a[i] = 1;
    }
    for (int i = 0; i < n - 2; ++i) {
        b[i] = 69;
    }

    initAndCall(a, b, n);

    for (int i = 0; i < n - 2; ++i)
      printf("%3d ", i + 1);
    printf("\n");
    for (int i = 0; i < n - 2; ++i)
        printf("%3d ", b[i]);
    printf("\n");

    free(a);
    free(b);
}