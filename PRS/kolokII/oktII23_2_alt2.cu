
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
int elements_per_reduction = BLOCK_SIZE * GRID_SIZE * 2;

__global__ void get_new_arr(int* a, int* b, int* c, int* n) {
    int tid_a = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid_a < *n * *n) {
        int row = tid_a / *n;
        int col = tid_a % *n;
        int tid_b = col * *n + row;
        
        c[tid_a] = min(a[tid_a], b[tid_b]);
        tid_a += blockDim.x * gridDim.x;
    }    
}

__host__ void get_matrix_c(int* a, int* b, int* c, int* n) {
    int* dev_a, *dev_b, *dev_c, *dev_n;
    hipMalloc((void**)&dev_a, *n * *n * sizeof(int));
    hipMalloc((void**)&dev_b, (*n * *n + 1) * sizeof(int));
    hipMalloc((void**)&dev_c, *n * *n * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, a, *n * *n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, *n * *n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    get_new_arr<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, dev_n);
    hipMemcpy(c, dev_c, *n * *n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_n);
}

__global__ void get_avgs(int* c, float* avgs, int n) {
    __shared__ float local_sum[BLOCK_SIZE];
    local_sum[threadIdx.x] = 0.0f;

    int tid = threadIdx.x * n + blockIdx.x;
    int col = blockIdx.x;
    while (col < n) {
        while (tid < n * n) {
            local_sum[threadIdx.x] += (float)c[tid] / n;
            tid += n * blockDim.x;
        }
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s)
                local_sum[threadIdx.x] += local_sum[threadIdx.x + s];
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            avgs[col] = local_sum[0];
        }     
        col += gridDim.x;
    }
}

__host__ void get_col_avgs(int* c, float* avgs, int n) {
    printf("N=%d\n", n);
    int* dev_a; float *dev_b;
    hipMalloc((void**)&dev_a, n * n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(float));
    hipMemcpy(dev_a, c, n * n * sizeof(int), hipMemcpyHostToDevice);

    get_avgs<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, n);
    hipMemcpy(avgs, dev_b, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
}

int main() {
    int N = 5;
    // scanf("%d", &N);
    int* a = (int*)malloc(N * N * sizeof(int));
    int* b = (int*)malloc(N * N * sizeof(int));
    int* c = (int*)malloc(N * N * sizeof(int));

    for (int i = 0; i < N * N; ++i) {
        a[i] = rand() % 50;
        b[i] = rand() % 50;
    }

    get_matrix_c(a, b, c, &N);
    float* avg_c = (float*)malloc(N * sizeof(float));
    get_col_avgs(c, avg_c, N);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%5.2f ", (float)c[i * N + j]);
        }
        printf("\n");
    }
    printf("--------------------------------------\n");
    for (int i = 0; i < N; ++i)
        printf("%5.2f ", avg_c[i]);
    printf("\n");

    free(a);
    free(b);
    free(c);
    free(avg_c);
}