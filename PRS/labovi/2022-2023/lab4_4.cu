#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define BLOCK_SIZE 32
#define GRID_SIZE 32

__global__ void kernel(int* A, int* B, int* n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= n[0])
        return;
    
    int s = 0;
    for (int i = 0; i < n[0]; ++i) {
        s += A[tid * n[0] + (i + threadIdx.x) % n[0]];
    }
    B[tid] = s;
}

__host__ void initAndCall(int* A, int* B, int* n) {
    int* dev_a, *dev_b, *dev_n;
    hipMalloc((void**)&dev_a, n[0] * n[0] * sizeof(int));
    hipMalloc((void**)&dev_b, n[0] * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, A, n[0] * n[0] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_n);

    hipMemcpy(B, dev_b, n[0] * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {
    int* n = (int*)malloc(sizeof(int));

    n[0] = 10;
    printf("%d\n", n[0]);

    int* A, *b;
    A = (int*)malloc(n[0] * n[0] * sizeof(int));
    b = (int*)malloc(n[0] * sizeof(int));

    int c = 0;
    for (int i = 0; i < n[0]; ++i) {
        for (int j = 0; j < n[0]; ++j) {
            A[i * n[0] + j] = i * n[0] + j;
        }
    }

    initAndCall(A, b, n);

    for (int i = 0; i < n[0]; ++i) {
        for (int j = 0; j < n[0]; ++j) {
            printf("%4d ", A[i * n[0] + j]);
        }
        printf(" | %4d\n", b[i]);
    }

    return 0;
}