#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

#define BLOCK_SIZE 32
#define GRID_SIZE 32

__global__ void kernel(int* a, int* b, int* c, int* n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    while (tid < n[0]) {
        c[tid] = a[tid] * b[tid];
        tid += n[0];
    }
}

__host__ void init_and_call(int* a, int* b, int* c, int* n) {
    int* dev_a, *dev_b, *dev_c, *dev_n;
    hipMalloc((void**)&dev_a, n[0] * sizeof(int));
    hipMalloc((void**)&dev_b, n[0] * sizeof(int));
    hipMalloc((void**)&dev_c, n[0] * sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMemcpy(dev_a, a, n[0] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n[0] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, dev_n);

    hipMemcpy(c, dev_c, n[0] * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {
    int* n = (int*)malloc(sizeof(int));

    n[0] = 10 + rand() % 100;
    int* a, *b, *c;
    a = (int*)malloc(n[0] * sizeof(int));
    b = (int*)malloc(n[0] * sizeof(int));
    c = (int*)malloc(n[0] * sizeof(int));

    for (int i = 0; i < n[0]; ++i) {
        a[i] = i;
        b[i] = i;
    }

    for (int i = 0; i < n[0]; ++i) {
        printf("%3d ", a[i]);
    }
    printf("\n");
    for (int i = 0; i < n[0]; ++i) {
        printf("%3d ", b[i]);
    }
    printf("\n");

    init_and_call(a, b, c, n);

    for (int i = 0; i < n[0]; ++i) {
        printf("%3d ", c[i]);
    }
    printf("\n");
    return 0;
}