#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <stdlib.h>

#define NUM_MAX 500.0f
#define BLOCK_SIZE 32
#define GRID_SIZE 32

__global__ void kernel(float* a, float* b, int* n) {
    __shared__ float local_block[BLOCK_SIZE];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid >= n[0] + 2)
        return;

    local_block[threadIdx.x] = a[tid];

    if (tid >= n[0])
        return;

    __syncthreads();

    float s = 0;
    s += local_block[threadIdx.x] * 3;
    s += local_block[threadIdx.x + 1] * 10;
    s += local_block[threadIdx.x + 1] * 7;
    s /= 20.0f;
    b[tid] = s;
}

__host__ void init_and_call(float* a, float* b, int* n) {
    float* dev_a, *dev_b; int *dev_n;
    hipMalloc((void**)&dev_a, (n[0] + 2) * sizeof(float));
    hipMalloc((void**)&dev_b, n[0] * sizeof(float));
    hipMalloc((void**)&dev_n, sizeof(int));

    hipMemcpy(dev_a, a, (n[0] + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<GRID_SIZE, BLOCK_SIZE>>>(dev_a, dev_b, dev_n);

    hipMemcpy(b, dev_b, n[0] * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_n);
}

int main() {
    int* n = (int*)malloc(sizeof(int));

    n[0] = 10 + rand() % 20;
    printf("%d\n", n[0]);

    float* a, *b;
    a = (float*)malloc((n[0] + 2) * sizeof(int));
    b = (float*)malloc(n[0] * sizeof(int));

    for (int i = 0; i < n[0] + 2; ++i) {
        a[i] = ((float)rand() / RAND_MAX) * NUM_MAX;
    }

    init_and_call(a, b, n);

    for (int i = 0; i < n[0] + 2; ++i) {
        printf("%7.2f ", a[i]);
    }
    printf("\n        ");

    for (int i = 0; i < n[0]; ++i) {
        printf("%7.2f ", b[i]);
    }
    printf("\n");
    
    return 0;
}